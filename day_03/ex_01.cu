
#include <hip/hip_runtime.h>
#include<stdio.h>
#define BS 8
#define N 10

void print(int *A,int n){
    for(int i=0; i<n; i++)
        printf("%d ",A[i]);
    printf("\n");
}

__global__ void add_array(int *A, int *B, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n) A[i] = A[i] + B[i];
}

int main(void){
    int threadsPerBlock, blocksPerGrid, n, *A, *B, *dA, *dB;
    n=N; threadsPerBlock=BS;
    blocksPerGrid = (n+BS-1)/BS;
    
    A = (int*) malloc(n*sizeof(int));
    B = (int*) malloc(n*sizeof(int));

    for(int i=0; i<n; i++) A[i]=i;
    for(int i=0; i<n; i++) B[i]=i*2;
    print(A, n);
    print(B, n);    

    hipMalloc((void**)&dA, n*sizeof(int));
    hipMalloc((void**)&dB, n*sizeof(int));

    hipMemcpy(dA, A, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, B, n*sizeof(int), hipMemcpyHostToDevice);

    add_array<<<blocksPerGrid,threadsPerBlock>>>(dA, dB, n);

    hipMemcpy(A , dA, n*sizeof(int), hipMemcpyDeviceToHost);

    print(A, n);
 
    hipFree(dA);
    free(A);

    return 0;
}