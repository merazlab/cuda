#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#define BS 8
#define N 10

void print(int *A,int n){
    for(int i=0; i<n; i++)
        printf("%d",A[i]);
}

__global__void increment(int *A,int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n) A[i]++;
}

int main(void){
    int threadsPerBlock, blocksPerGrid, n, *A, *dA;
    n=N; threadsPerBlock=BS;
    blocksPerGrid = (n+BS-1)/BS;
    A=(int*) malloc(n*sizeof(int));
    for(int i=0; i<n; i++) A[i]=i*10;
    //0 10 20 30 40 50 60 70 80 90 
    hipMalloc((void**)&dA,n*sizeof(int));
    hipMemcpy(dA,A,n*sizeof(int),hipMemcpyHostToDevice);
    increment<<<blocksPerGrid,threadsPerBlock>>>(dA,n);
    hipMemcpy(A,dA,n*sizeof(int),hipMemcpyDeviceToHost);
    print(A, n);
    //1 11 21 31 41 51 61 71 81 91 
    hipFree(dA);
    free(A);
    return 0;
}